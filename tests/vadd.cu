// dot_product.cu
// nvcc dot_product.cu -o dot_product


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define BLOCK_SIZE 32

const int DSIZE = 256;
const int a = 1;
const int b = 1;

// error checking macro
#define cudaCheckErrors(msg)                                    \
        do {                                                        \
                hipError_t __err = hipGetLastError();                 \
                if (__err != hipSuccess) {                             \
                        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n",  \
                                        msg, hipGetErrorString(__err),             \
                                        __FILE__, __LINE__);                        \
                        fprintf(stderr, "*** FAILED - ABORTING***\n");      \
                        exit(1);                                            \
                }                                                       \
        } while (0)


// CUDA kernel that runs on the GPU
__global__ void dot_product(const int *A, const int *B, int *C, int N) {

        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < N) {
                printf("Adding %i \n", A[idx] * B[idx]);
                atomicAdd(C, A[idx] * B[idx]);
        }

}


int main() {

        // Create the device and host pointers
        int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

        // Fill in the host pointers 
        h_A = new int[DSIZE];
        h_B = new int[DSIZE];
        h_C = new int;
        for (int i = 0; i < DSIZE; i++){
                h_A[i] = a;
                h_B[i] = b;
        }

        *h_C = 0;

        // Allocate device memory 
        hipMalloc(&d_A, DSIZE*sizeof(int));
        hipMalloc(&d_B, DSIZE*sizeof(int));
        hipMalloc(&d_C, sizeof(int));

        // Check memory allocation for errors
        cudaCheckErrors("Was memory allocation successful");

        // Copy the vectors on GPU
        hipMemcpy(d_A, h_A, DSIZE*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, DSIZE*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_C, h_C, sizeof(int),       hipMemcpyHostToDevice);

        // Check memory copy for errors
        cudaCheckErrors("Was memory copy from host to device successful");

        // Define block/grid dimensions and launch kernel
        int grid_size = (DSIZE/BLOCK_SIZE);
        // myKernel<<<nBlocks, nThreads>>>
        dot_product<<<grid_size, BLOCK_SIZE>>>(d_A, d_B, d_C, DSIZE);

        // Copy results back to host
        hipMemcpy(h_C, d_C, sizeof(int), hipMemcpyDeviceToHost);

        // Check copy for errors
        cudaCheckErrors("Was memory copy from device to host successful");

        // Verify result
        printf("After kernel: value of h_C is %i\n", *h_C);

        // Free allocated memory
        free(h_A);
        free(h_B);
        free(h_C);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);

        return 0;
}
